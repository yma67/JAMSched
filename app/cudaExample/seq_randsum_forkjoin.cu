//
// Created by mayuxiang on 2020-11-21.
//
#include <cstdint>
#include <random>
#include <chrono>
#include <iostream>
#include <thread>
#include <vector>
#include <cassert>
#include "hip/hip_runtime.h"
#include "deps/Kernel.h"

constexpr bool useThread = false;
constexpr int kNumTrails = 256;
constexpr int kPerDimLen = 256;
constexpr int kNumIteration = 8;

static std::tuple<hipStream_t, int*, int*, int*, int*, int*, int*> Compute() {
    int *host_a, *host_b, *host_c, *dev_a, *dev_b, *dev_c;
    hipStream_t stream;
    hipStreamCreate(&stream);
    auto res1 = hipHostAlloc(&host_a, kPerDimLen * kPerDimLen * kNumIteration * sizeof(int), hipHostMallocDefault);
    auto res2 = hipHostAlloc(&host_b, kPerDimLen * kPerDimLen * kNumIteration * sizeof(int), hipHostMallocDefault);
    auto res3 = hipHostAlloc(&host_c, kPerDimLen * kPerDimLen * kNumIteration * sizeof(int), hipHostMallocDefault);
    if (res1 != hipSuccess) {
        printf("hostAlloc Error 1\n");
        return std::make_tuple(stream, host_a, host_b, host_c, dev_a, dev_b, dev_c);
    }
    if (res2 != hipSuccess) {
        printf("hostAlloc Error 3\n");
        return std::make_tuple(stream, host_a, host_b, host_c, dev_a, dev_b, dev_c);
    }
    if (res3 != hipSuccess) {
        printf("hostAlloc Error 3\n");
        return std::make_tuple(stream, host_a, host_b, host_c, dev_a, dev_b, dev_c);;
    }
    hipMalloc((void**)(&dev_a), kPerDimLen * kPerDimLen * sizeof( int) );
    hipMalloc((void**)(&dev_b), kPerDimLen * kPerDimLen * sizeof( int) );
    hipMalloc((void**)(&dev_c), kPerDimLen * kPerDimLen * sizeof( int) );
    auto result = GetRandomArray(host_a, host_b, kPerDimLen * kPerDimLen, kPerDimLen * kPerDimLen * kNumIteration);
    for ( int i = 0; i < kPerDimLen * kPerDimLen * kNumIteration; i += kPerDimLen * kPerDimLen) {
        hipMemcpyAsync( dev_a, host_a + i, kPerDimLen * kPerDimLen * sizeof( int), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync( dev_b, host_b + i, kPerDimLen * kPerDimLen * sizeof( int), hipMemcpyHostToDevice, stream);
        CircularSubarrayInnerProduct<<<kPerDimLen * kPerDimLen / 256, 256, 0, stream>>>(dev_a, dev_b, dev_c, kPerDimLen * kPerDimLen);
        hipMemcpyAsync( host_c + i, dev_c, kPerDimLen * kPerDimLen * sizeof( int), hipMemcpyDeviceToHost, stream);
    }
    return std::make_tuple(stream, host_a, host_b, host_c, dev_a, dev_b, dev_c);
}

static void Collect(std::tuple<hipStream_t, int*, int*, int*, int*, int*, int*> p) {
    auto& [stream, host_a, host_b, host_c, dev_a, dev_b, dev_c] = p;
    hipStreamSynchronize(stream);
    for (int i = 0; i < kPerDimLen * kPerDimLen * kNumIteration; i++) assert(result[i] == host_c[i]);
    hipFree(dev_a);
    hipFree( dev_b);
    hipFree( dev_c);
    hipHostFree(host_a);
    hipHostFree(host_b);
    hipHostFree(host_c);
    hipStreamDestroy(stream);
}

int main() {
    std::vector<std::thread> px;
    std::vector<std::tuple<hipStream_t, int*, int*, int*, int*, int*, int*>> sx;
    auto startCuda = std::chrono::high_resolution_clock::now();
    InitDummy();
    for (int i = 0; i < kNumTrails; i++) {
        if constexpr(useThread) {
            px.emplace_back(Compute);
        } else {
            sx.push_back(Compute());
        }
    }
    if constexpr(useThread) {
        for (auto& p: px) p.join();   
    } else {
        for (auto& s: sx) Collect(s);  
    }
    auto dur = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::high_resolution_clock::now() - startCuda).count();
    std::cout << "CPU time: " << dur << " us" << std::endl;
    return 0;
}
