//
// Created by mayuxiang on 2020-11-21.
//
#include <cstdint>
#include <random>
#include <chrono>
#include <iostream>
#include <vector>
#include <cmath>
#include <cassert>
#include "hip/hip_runtime.h"
#include "deps/Kernel.h"

constexpr int kSize { 256 * 256 * 256 * 8 };

struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }
    void Start();
    void Stop();
    float Elapsed();
};

inline void GpuTimer::Start()
{
    hipEventRecord(start, 0);
}

inline void GpuTimer::Stop()
{
    hipEventRecord(stop, 0);
}

inline float GpuTimer::Elapsed()
{
    float elapsed;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);
    return elapsed;
}

static void Compute() {
    int *host_a, *host_b, *host_c, *dev_a, *dev_b, *dev_c;
    auto res1 = hipHostAlloc(&host_a, kSize * sizeof(int), hipHostMallocDefault);
    auto res2 = hipHostAlloc(&host_b, kSize * sizeof(int), hipHostMallocDefault);
    auto res3 = hipHostAlloc(&host_c, kSize * sizeof(int), hipHostMallocDefault);
    if (res1 != hipSuccess) {
        printf("hostAlloc Error 1\n");
        return;
    }
    if (res2 != hipSuccess) {
        printf("hostAlloc Error 3\n");
        return;
    }
    if (res3 != hipSuccess) {
        printf("hostAlloc Error 3\n");
        return;
    }
    hipMalloc((void**)(&dev_a), kSize * sizeof( int) );
    hipMalloc((void**)(&dev_b), kSize * sizeof( int) );
    hipMalloc((void**)(&dev_c), kSize * sizeof( int) );
    auto result = GetRandomArray(host_a, host_b, kSize, kSize);
    hipMemcpy( dev_a, host_a, kSize * sizeof( int), hipMemcpyHostToDevice);
    hipMemcpy( dev_b, host_b, kSize * sizeof( int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    GpuTimer t;
    t.Start();
    auto startCuda = std::chrono::high_resolution_clock::now();
    CircularSubarrayInnerProduct<<<kSize / 256, 256>>>(dev_a, dev_b, dev_c, kSize);
    /*for (int i = 0; i < kSize; i++) {
        host_c[i] = 0;
        for (int j = i; j < i + 128; j++) {
            host_c[i] += host_a[j % kSize] * host_b[j % kSize];
        }
    }*/
    hipDeviceSynchronize();
    auto dur = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::high_resolution_clock::now() - startCuda).count();
    t.Stop();
    hipMemcpy( host_c, dev_c, kSize * sizeof( int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    for (long i = 0; i < kSize; i++) assert(result[i] == host_c[i]);
    std::cout << "Kernel GPU time: " << t.Elapsed() << " us" << std::endl;
    std::cout << "Kernel CPU time: " << dur << " us" << std::endl;
    hipFree(dev_a);
    hipFree( dev_b);
    hipFree( dev_c);
    hipHostFree(host_a);
    hipHostFree(host_b);
    hipHostFree(host_c);
}

int main() {
    auto startCuda = std::chrono::high_resolution_clock::now();
    int dn;
    hipGetDevice(&dn);
    Compute();
    auto dur = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::high_resolution_clock::now() - startCuda).count();
    std::cout << "CPU time: " << dur << " us" << std::endl;
    return 0;
}
