//
// Created by mayuxiang on 2020-11-21.
//
#include <cstdint>
#include <random>
#include <chrono>
#include <iostream>
#include <thread>
#include <vector>
#include <cassert>
#include "hip/hip_runtime.h"

constexpr bool useThread = true;
constexpr int kNumTrails = 128;
constexpr size_t kPerDimLen = 256;
constexpr size_t kNumIteration = 8;

__global__
void vector_add( int * a, int * b, int * c, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int lim = idx + 256;
    if (lim > size) lim = size;
    for (int i = idx; i < lim; i++) c[idx] += a[i] * b[i];
    if (idx < size) {
        c[idx] = a[idx] + b[idx];
    }
}

static void Compute() {
    int *host_a, *host_b, *host_c, *dev_a, *dev_b, *dev_c;
    std::vector<int> result;
	hipStream_t stream;
	hipStreamCreate(&stream);
	auto res1 = hipHostAlloc(&host_a, kPerDimLen * kPerDimLen * kNumIteration * sizeof(int), hipHostMallocDefault);
	auto res2 = hipHostAlloc(&host_b, kPerDimLen * kPerDimLen * kNumIteration * sizeof(int), hipHostMallocDefault);
	auto res3 = hipHostAlloc(&host_c, kPerDimLen * kPerDimLen * kNumIteration * sizeof(int), hipHostMallocDefault);
	if (res1 != hipSuccess) {
	    printf("shut up 1\n");
	    return;
	}
	if (res2 != hipSuccess) {
	    printf("shut up 3\n");
	    return;
	}
	if (res3 != hipSuccess) {
	    printf("shut up 3\n");
	    return;
	}
	hipMalloc((void**)(&dev_a), kPerDimLen * kPerDimLen * sizeof( int) );
	hipMalloc((void**)(&dev_b), kPerDimLen * kPerDimLen * sizeof( int) );
	hipMalloc((void**)(&dev_c), kPerDimLen * kPerDimLen * sizeof( int) );
	{
	    std::minstd_rand generator;
	    std::uniform_int_distribution<> distribution(1, 6);
	    for ( int i = 0; i < kPerDimLen * kPerDimLen * kNumIteration; ++i) {
		host_a[i] = distribution(generator);
		host_b[i] = distribution(generator);
		result.push_back(host_a[i] + host_b[i]);
	    }
	}
	for ( int i = 0; i < kPerDimLen * kPerDimLen * kNumIteration; i += kPerDimLen * kPerDimLen) {
	    hipMemcpyAsync( dev_a, host_a + i, kPerDimLen * kPerDimLen * sizeof( int), hipMemcpyHostToDevice, stream);
	    hipMemcpyAsync( dev_b, host_b + i, kPerDimLen * kPerDimLen * sizeof( int), hipMemcpyHostToDevice, stream);
	    vector_add<<<kPerDimLen * kPerDimLen / 256, 256, 0, stream>>>(dev_a, dev_b, dev_c, kPerDimLen * kPerDimLen);
	    hipMemcpyAsync( host_c + i, dev_c, kPerDimLen * kPerDimLen * sizeof( int), hipMemcpyDeviceToHost, stream);
	}
	hipStreamSynchronize(stream);
	for (int i = 0; i < kPerDimLen * kPerDimLen * kNumIteration; i++) assert(result[i] == host_c[i]);
	hipFree(dev_a);
	hipFree( dev_b);
	hipFree( dev_c);
	hipHostFree(host_a);
	hipHostFree(host_b);
	hipHostFree(host_c);
	hipStreamDestroy(stream);
}

int main() {
    std::vector<std::thread> px;
    auto startCuda = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < kNumTrails; i++) {
        if constexpr(useThread) {
            px.emplace_back(Compute);
        } else {
            Compute();
        }
    }
    if constexpr(useThread) {
         for (auto& p: px) p.join();   
    }
    auto dur = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::high_resolution_clock::now() - startCuda).count();
    std::cout << "CPU time: " << dur << " us" << std::endl;
    return 0;
}
