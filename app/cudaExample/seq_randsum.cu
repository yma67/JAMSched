//
// Created by mayuxiang on 2020-11-21.
//
#include <cstdint>
#include <random>
#include <chrono>
#include <iostream>
#include <thread>
#include <vector>
#include <cassert>
#include "hip/hip_runtime.h"
#include "deps/Kernel.h"

constexpr bool useThread = true;
constexpr int kNumTrails = 256;
constexpr int kPerDimLen = 256;
constexpr int kNumIteration = 8;

static void Compute() {
    int *host_a, *host_b, *host_c, *dev_a, *dev_b, *dev_c;
    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    auto res1 = hipHostAlloc(&host_a, kPerDimLen * kPerDimLen * kNumIteration * sizeof(int), hipHostMallocDefault);
    auto res2 = hipHostAlloc(&host_b, kPerDimLen * kPerDimLen * kNumIteration * sizeof(int), hipHostMallocDefault);
    auto res3 = hipHostAlloc(&host_c, kPerDimLen * kPerDimLen * kNumIteration * sizeof(int), hipHostMallocDefault);
    if (res1 != hipSuccess) {
        printf("hostAlloc Error 1\n");
        return;
    }
    if (res2 != hipSuccess) {
        printf("hostAlloc Error 3\n");
        return;
    }
    if (res3 != hipSuccess) {
        printf("hostAlloc Error 3\n");
        return;
    }
    hipMalloc((void**)(&dev_a), kPerDimLen * kPerDimLen * sizeof( int) );
    hipMalloc((void**)(&dev_b), kPerDimLen * kPerDimLen * sizeof( int) );
    hipMalloc((void**)(&dev_c), kPerDimLen * kPerDimLen * sizeof( int) );
    auto result = GetRandomArray(host_a, host_b, kPerDimLen * kPerDimLen, kPerDimLen * kPerDimLen * kNumIteration);
    for ( int i = 0; i < kPerDimLen * kPerDimLen * kNumIteration; i += kPerDimLen * kPerDimLen) {
        hipMemcpyAsync( dev_a, host_a + i, kPerDimLen * kPerDimLen * sizeof( int), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync( dev_b, host_b + i, kPerDimLen * kPerDimLen * sizeof( int), hipMemcpyHostToDevice, stream);
        CircularSubarrayInnerProduct<<<kPerDimLen * kPerDimLen / 256, 256, 0, stream>>>(dev_a, dev_b, dev_c, kPerDimLen * kPerDimLen);
        hipMemcpyAsync( host_c + i, dev_c, kPerDimLen * kPerDimLen * sizeof( int), hipMemcpyDeviceToHost, stream);
    }
    hipStreamSynchronize(stream);
    for (int i = 0; i < kPerDimLen * kPerDimLen * kNumIteration; i++) assert(result[i] == host_c[i]);
    hipFree(dev_a);
    hipFree( dev_b);
    hipFree( dev_c);
    hipHostFree(host_a);
    hipHostFree(host_b);
    hipHostFree(host_c);
    hipStreamDestroy(stream);
}

int main() {
    std::vector<std::thread> px;
    auto startCuda = std::chrono::high_resolution_clock::now();
    InitDummy();
    for (int i = 0; i < kNumTrails; i++) {
        if constexpr(useThread) {
            px.emplace_back(Compute);
        } else {
            Compute();
        }
    }
    if constexpr(useThread) {
         for (auto& p: px) p.join();   
    }
    auto dur = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::high_resolution_clock::now() - startCuda).count();
    std::cout << "CPU time: " << dur << " us" << std::endl;
    return 0;
}
