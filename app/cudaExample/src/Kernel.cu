#include "hip/hip_runtime.h"
//
// Created by mayuxiang on 2020-11-11.
//
#include "../deps/Kernel.h"
#include "../deps/LogicalInput.h"
#include "../deps/cnmem.h"
#include <vector>
#include "jamc-cuda.h"
#include "hip/hip_runtime.h"
#include <chrono>
#include <tuple>
#include <array>
#include <algorithm>
#include <random>
#include <memory>
#include <cassert>
#include <functional>
#include <nvToolsExt.h>
#include <nvToolsExtCuda.h>


template <typename... Args>
class CommandArgs {
    using ArgTupleType = std::tuple<Args...>;
    std::array<void*, std::tuple_size<ArgTupleType>::value> arrayArgs;
    ArgTupleType actArgs;
public:
    CommandArgs() = default;
    CommandArgs(Args... args) : actArgs(std::forward<Args>(args)...) {
        std::apply([this](auto& ...xs) { arrayArgs = {(&xs)...}; }, actArgs);
    }
    void** GetCudaKernelArgs() { return arrayArgs.data(); }
};

constexpr int kInnerProductSize = 128;

__global__
void CircularSubarrayInnerProduct( int * a, int * b, int * c, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    c[idx] = 0;
    for (int i = idx; i < idx + kInnerProductSize; i++) {
        c[idx] += a[i % size] * b[i % size];
    }
}

void InitDummy()
{
    hipStream_t dummys;
    hipStreamCreate(&dummys);
    hipStreamDestroy(dummys);
}

std::vector<int> GetRandomArray(int * ha, int * hb, int sz, int fsz)
{
    assert(kInnerProductSize > sz);
    nvtxRangeId_t id2 = nvtxRangeStart("GetRandomArray");
    std::vector<int> res(fsz, 0);
    std::minstd_rand generator;
    std::uniform_int_distribution<> distribution(1, 25);
    for ( int i = 0; i < fsz; i += sz) {
        int cumu = 0;
        for (int j = 0; j < sz; j++) {
            ha[i + j] = distribution(generator);
            hb[i + j] = distribution(generator);
        }
        for (int j = 0; j < kInnerProductSize; j++) {
            cumu += ha[i + j] * hb[i + j];
        }
        for (int j = 0; j < sz; j++) {
            res[i + j] = cumu;
            cumu -= ha[i + j] * hb[i + j];
            cumu += ha[i + ((j + kInnerProductSize) % sz)] * hb[i + ((j + kInnerProductSize) % sz)];
        }
    }
    nvtxRangeEnd(id2);
    return res;
}

void KernelInvoker(hipStream_t stream, int* host_a, int* host_b, int* host_c, int* dev_a, int* dev_b, int* dev_c, int size, int numIteration)
{
    int full_size = numIteration * size;
    auto result = GetRandomArray(host_a, host_b, size, full_size);
    auto args = std::make_unique<CommandArgs<int*, int*, int*, int>>(dev_a, dev_b, dev_c, size);
    for ( int i = 0; i < full_size; i += size) {
        hipMemcpyAsync( dev_a, host_a + i, size * sizeof( int), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync( dev_b, host_b + i, size * sizeof( int), hipMemcpyHostToDevice, stream);
        hipLaunchKernel((void*)CircularSubarrayInnerProduct, dim3(size / 256), dim3(256), args->GetCudaKernelArgs(), 0, stream);
        hipMemcpyAsync( host_c + i, dev_c, size * sizeof( int), hipMemcpyDeviceToHost, stream);
    }
    WaitForCudaStream(stream);
    for (int i = 0; i < full_size; i++) assert(result[i] == host_c[i]);
}
